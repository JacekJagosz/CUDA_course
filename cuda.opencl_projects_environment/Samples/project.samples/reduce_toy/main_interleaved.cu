#include <iostream>
#include <ctime>
#include <stdlib.h>

#include "hip/hip_runtime.h"

//#include "utils.h"

//#include "reduce.h"

__global__ void reduceinter(unsigned int* outdata, unsigned int* indata, unsigned int len) {
	extern __shared__ unsigned int sdata[];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = 0;

	if (i < len)
	{
		sdata[tid] = indata[i];
	}

	__syncthreads();

	// Interleaved addressing causes significant thread divergence
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) 
		outdata[blockIdx.x] = sdata[0];
}

void generate_input(unsigned int* input, unsigned int input_len)
{
	for (unsigned int i = 0; i < input_len; ++i)
	{
		input[i] = i;
	}
}

unsigned int cpu_simple_sum(unsigned int* h_in, unsigned int h_in_len)
{
	unsigned int total_sum = 0;

	for (unsigned int i = 0; i < h_in_len; ++i)
	{
		total_sum = total_sum + h_in[i];
	}

	return total_sum;
}

int main()
{
	std::cout<< "hello" <<std::endl;
	return;
}
