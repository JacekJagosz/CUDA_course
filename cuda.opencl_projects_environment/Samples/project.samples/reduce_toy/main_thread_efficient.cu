#include <iostream>
#include <ctime>
#include <stdlib.h>

#include "hip/hip_runtime.h"

//#include "utils.h"

//#include "reduce.h"

__global__ void reduce3(unsigned int* outdata, unsigned int* indata, unsigned int len) {
	extern __shared__ unsigned int sdata[];

	// each thread loads one element from global to shared mem
	// Do the first stage of the reduction on the global-to-shared load step
	// This reduces the previous inefficiency of having half of the threads being
	//  inactive on the first for-loop iteration below (previous first step of reduction)
	// Previously, only less than or equal to 512 out of 1024 threads in a block are active.
	// Now, all 512 threads in a block are active from the start
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdata[tid] = 0;

	if (i < len)
	{
		sdata[tid] = indata[i] + indata[i + blockDim.x];
	}

	__syncthreads();

	// do reduction in shared mem
	// this loop now starts with s = 512 / 2 = 256
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		outdata[blockIdx.x] = sdata[0];
}

void generate_input(unsigned int* input, unsigned int input_len)
{
	for (unsigned int i = 0; i < input_len; ++i)
	{
		input[i] = i;
	}
}

unsigned int cpu_simple_sum(unsigned int* h_in, unsigned int h_in_len)
{
	unsigned int total_sum = 0;

	for (unsigned int i = 0; i < h_in_len; ++i)
	{
		total_sum = total_sum + h_in[i];
	}

	return total_sum;
}

int main()
{
	std::cout<< "hello" <<std::endl;
	return;
}
